#include "hip/hip_runtime.h"
# include<stdio.h>
# include<stdlib.h>
# include<math.h>
# include<hip/hip_runtime.h>

__global__ static void sum_of_e(double *result, int power_of_e, int n, int thread_num, int block_num){
	double temp = 0.0;
	const int thread_id = threadIdx.x;
	const int block_id = blockIdx.x;
	double f;

	for (int i = block_id * thread_num + thread_id; i <= n; i += block_num * thread_num){
		f = 1.0;
		for (int j = 2; j <= i; j++){
			f *= j;
		}
		temp += pow(power_of_e, i) / f;
	}
	result[block_id * thread_num + thread_id] = temp;
}

int main(int argc, char *argv[]){
	int block_num = atoi(argv[1]);
	int thread_num = atoi(argv[2]);	
	int power_of_e = atoi(argv[3]);
	int n = atoi(argv[4]);

	double result = exp(power_of_e);
	printf("   Real              value   : %.50f\n", result);
	
	double result_cpu = 1.0;
	double f;
	clock_t start_cpu = clock();
	for (int i = 1; i <= n; i++){
		f = 1.0;
		for (int j = 2; j <= i; j++){
			f *= j;
		}
		result_cpu += pow(power_of_e, i) / f;
	}
	clock_t end_cpu = clock();

	printf("CPU time: %Lf, CPU value: %.50f\n", (long double)(end_cpu - start_cpu) / CLOCKS_PER_SEC, result_cpu);

	double *result_gpu;
	hipMalloc((void**) &result_gpu, sizeof(double) * thread_num * block_num);
	clock_t start_gpu = clock();
	sum_of_e<<<block_num, thread_num>>>(result_gpu, power_of_e, n, thread_num, block_num);
	clock_t end_gpu = clock();
	
	double result_of_gpu[thread_num * block_num];
	hipMemcpy(&result_of_gpu, result_gpu, sizeof(double) * thread_num * block_num, hipMemcpyDeviceToHost);
	hipFree(result_gpu);

	double final_e = 0.0;
	for (int i = 0; i < thread_num * block_num; i++){
		final_e += result_of_gpu[i];
	}

	printf("GPU time: %Lf, GPU value: %.50f\n", (long double)(end_gpu - start_gpu) / CLOCKS_PER_SEC, final_e);

	return 0;
}
